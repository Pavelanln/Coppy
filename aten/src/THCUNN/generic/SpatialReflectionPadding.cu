
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SpatialReflectionPadding.cu"
#else

void THNN_(SpatialReflectionPadding_updateOutput)(THCState *state,
           THCTensor *input,
           THCTensor *output,
           int64_t padL, int64_t padR,
           int64_t padT, int64_t padB) {
  THArgCheck(THCTensor_canUse32BitIndexMath(state, input), 2,
             "input tensor must fit into 32-bit index math");

  int64_t planeDim = 0;
  int64_t dimh = 1;
  int64_t dimw = 2;
  int64_t numBatch = 1;

  int64_t numInputDims = THCTensor_(nDimension)(state, input);
  THCUNN_argCheck(state, !input->is_empty() && (numInputDims == 3 || numInputDims == 4), 2, input,
                  "non-empty 3D or 4D (batch mode) tensor expected for input, but got: %s")

  if (numInputDims == 4) {
    numBatch = THCTensor_(size)(state, input, 0);
    planeDim++;
    dimh++;
    dimw++;
  }

  int64_t numPlanes = THCTensor_(size)(state, input, planeDim);
  int64_t inputH = THCTensor_(size)(state, input, dimh);
  int64_t inputW = THCTensor_(size)(state, input, dimw);

  THArgCheck(padL < inputW && padR < inputW, 4,
             "Padding size should be less than the corresponding input dimension, "
             "but got: padding (%d, %d) at dimension %d of input %s",
             padL, padR, dimw, THCTensor_(sizeDesc)(state, input).str);

  THArgCheck(padT < inputH && padB < inputH, 6,
             "Padding size should be less than the corresponding input dimension, "
             "but got: padding (%d, %d) at dimension %d of input %s",
             padT, padB, dimh, THCTensor_(sizeDesc)(state, input).str);

  int64_t outputH = inputH + padT + padB;
  int64_t outputW  = inputW + padL + padR;

  THArgCheck(outputW >= 1 || outputH >= 1, 2,
             "input (H: %d, W: %d)is too small."
             " Calculated output H: %d W: %d",
             inputH, inputW, outputH, outputW);

  THCDeviceTensor<real, 4> devInput;
  THCDeviceTensor<real, 4> devOutput;

  if (numInputDims == 3) {
    THCTensor_(resize3d)(state, output, numPlanes, outputH, outputW);

    devInput = toDeviceTensor<real, 3>(state, input).upcastOuter<4>();
    devOutput = toDeviceTensor<real, 3>(state, output).upcastOuter<4>();
  } else {
    THCTensor_(resize4d)(state, output, numBatch, numPlanes, outputH, outputW);

    devInput = toDeviceTensor<real, 4>(state, input);
    devOutput = toDeviceTensor<real, 4>(state, output);
  }

  int64_t outputPlaneSize = devOutput.getSize(2) * devOutput.getSize(3);
  dim3 gridSize(THCCeilDiv(outputPlaneSize, 256),
            devOutput.getSize(1),
            devOutput.getSize(0));
  dim3 blockSize(outputPlaneSize > 256 ? 256 : outputPlaneSize);

  SpatialReflectionPadding_updateOutput<<<gridSize, blockSize, 0, THCState_getCurrentStream(state)>>>(
    devInput, devOutput, padT, padB, padL, padR);
  THCudaCheck(cudaGetLastError());
}

void THNN_(SpatialReflectionPadding_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int64_t padL, int64_t padR,
           int64_t padT, int64_t padB) {

  THArgCheck(THCTensor_canUse32BitIndexMath(state, input), 2,
                "input tensor must fit into 32-bit index math");
  THArgCheck(THCTensor_canUse32BitIndexMath(state, gradOutput), 3,
                "output gradient tensor must fit into 32-bit index math");

  int64_t planeDim = 0;
  int64_t dimh = 1;
  int64_t dimw = 2;

  int64_t numInputDims = THCTensor_(nDimension)(state, input);
  if (numInputDims == 4) {
    planeDim++;
    dimh++;
    dimw++;
  }
  int64_t iheight = input->size[dimh];
  int64_t iwidth = input->size[dimw];
  int64_t oheight = iheight + padT + padB;
  int64_t owidth  = iwidth + padL + padR;

  THArgCheck(owidth == THCTensor_(size)(state, gradOutput, dimw), 3,
             "gradOutput width unexpected. Expected: %d, Got: %d",
             owidth, THCTensor_(size)(state, gradOutput, dimw));
  THArgCheck(oheight == THCTensor_(size)(state, gradOutput, dimh), 3,
             "gradOutput height unexpected. Expected: %d, Got: %d",
             oheight, THCTensor_(size)(state, gradOutput, dimh));

  THCTensor_(resizeAs)(state, gradInput, input);
  THCTensor_(zero)(state, gradInput);

  THCDeviceTensor<real, 4> devGradInput;
  THCDeviceTensor<real, 4> devGradOutput;

  if (numInputDims == 3) {
    devGradInput = toDeviceTensor<real, 3>(state, gradInput).upcastOuter<4>();
    devGradOutput = toDeviceTensor<real, 3>(state, gradOutput).upcastOuter<4>();
  } else {
    devGradInput = toDeviceTensor<real, 4>(state, gradInput);
    devGradOutput = toDeviceTensor<real, 4>(state, gradOutput);
  }

  int64_t outputPlaneSize = devGradOutput.getSize(2) * devGradOutput.getSize(3);
  dim3 gridSize(THCCeilDiv(outputPlaneSize, 256),
            devGradOutput.getSize(1),
            devGradOutput.getSize(0));
  dim3 blockSize(outputPlaneSize > 256 ? 256 : outputPlaneSize);

  SpatialReflectionPadding_updateGradInput<<<gridSize, blockSize, 0, THCState_getCurrentStream(state)>>>(
    devGradInput, devGradOutput, padT, padB, padL, padR);
  THCudaCheck(cudaGetLastError());
}

#endif
